#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
// #define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&value)[0])
#define HALF2(value) (reinterpret_cast<__half2*>(&value)[0])
#define LD128(value) (reinterpret_cast<float4*>(&value)[0])


#define CHECK_TORCH_TYPE(tensor, type)                      \
if(tensor.options().dtype() != type){                       \
    throw std::runtime_error("tensor type must be"#type);   \
}

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f16_f32(half val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return __half2float(val);
}

template<const int THREAD_NUM>
__device__ __forceinline__ float block_reduce_sum_f32_f32(float val) {
    __shared__ float shm[THREAD_NUM / WARP_SIZE];
    val = __expf(val);
    int warp_idx = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    val = warp_reduce_sum_f32<32>(val);
    if(lane == 0)
        shm[warp_idx] = val;
    __syncthreads();
    if(threadIdx.x < 32)
        val = warp_reduce_sum_f32<32>(lane < (THREAD_NUM / WARP_SIZE)? shm[lane] : 0.0f);
    return val;
}

template<const int THREAD_NUM>
__global__ void softmax_kernel_f32(float* a, float* b, float* total, int N){
    int tid = threadIdx.x + THREAD_NUM * blockIdx.x;
    float reg = a[tid];
    float sum = block_reduce_sum_f32_f32<THREAD_NUM>(reg);
    if(threadIdx.x == 0){
        atomicAdd(total, sum);
    }
    __syncthreads();
    b[tid] = __expf(reg) / total[0];
}


#define TORCH_BINDING_SOFTMAX(TYPE_INPUT, TYPE_TORCH, ELEMENT_TYPE, ACCESS_NUM)   \
void softmax_##TYPE_INPUT(torch::Tensor a, torch::Tensor b){             \
    CHECK_TORCH_TYPE(a, TYPE_TORCH); \
    CHECK_TORCH_TYPE(b, TYPE_TORCH); \
    auto options = torch::TensorOptions().dtype((TYPE_TORCH)).device(torch::kCUDA, 0);\
    auto total = torch::zeros({1}, options);                                       \
    const int n = a.dim();          \
    int N = 1; \
    for(int i = 0; i < n; i++){ \
        N *= a.size(i); \
    } \
    const int thread_num = 256; \
    dim3 block(thread_num); \
    dim3 grid((N + thread_num - 1) / thread_num / ACCESS_NUM); \
    softmax_kernel_##TYPE_INPUT<thread_num><<<grid, block>>>( \
        reinterpret_cast<ELEMENT_TYPE*>(a.data_ptr()), \
        reinterpret_cast<ELEMENT_TYPE*>(b.data_ptr()), \
        reinterpret_cast<ELEMENT_TYPE*>(total.data_ptr()), \
        N \
    ); \
}

TORCH_BINDING_SOFTMAX(f32,    torch::kFloat32,       float,              1)
// TORCH_BINDING_SOFTMAX(f32x4,  torch::kFloat32,       float,              4)


#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
m.def(STRINGFY(func), func);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(softmax_f32)
//   TORCH_BINDING_COMMON_EXTENSION(dot_prod_f32_f32x4)
//   TORCH_BINDING_COMMON_EXTENSION(dot_prod_f16_f16)
//   TORCH_BINDING_COMMON_EXTENSION(dot_prod_f16_f16x8)
}