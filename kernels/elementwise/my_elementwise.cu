#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
// #define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&value)[0])
#define HALF2(value) (reinterpret_cast<__half2*>(&value)[0])
#define LD128(value) (reinterpret_cast<float4*>(&value)[0])

#define CHECK_TORCH_TYPE(tensor, type)                      \
if(tensor.options().dtype() != type){                       \
    throw std::runtime_error("tensor type must be"#type);   \
}


__global__ void elementwise_add_kernel_f16(half* a, half* b, half* c, int N){
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    c[idx] = b[idx] + a[idx];
}


__global__ void elementwise_add_kernel_f16x2(half* a, half* b, half* c, int N){
    int idx = (threadIdx.x + (blockIdx.x * blockDim.x)) * 2;
    half2 res = __hadd2(HALF2(a[idx]), HALF2(b[idx]));
    c[idx] = res.x;
    c[idx + 1] = res.y;
}


__global__ void elementwise_add_kernel_f16x8(half* a, half* b, half* c, int N){
    int idx = (threadIdx.x + (blockIdx.x * blockDim.x)) * 8;
    half reg_a[8], reg_b[8], reg_c[8];
    LD128(reg_a[0]) = LD128(a[idx]);
    LD128(reg_b[0]) = LD128(b[idx]);
    #pragma unroll
    for(int i = 0; i < 8; i += 2){
        HALF2(reg_c[i]) = __hadd2(HALF2(reg_a[i]), HALF2(reg_b[i]));
    }
    LD128(c[idx]) = LD128(reg_c[0]);
}


#define ELEMENT_WISE_TORCH_BINDING(INPUT_TYPE, TORCH_TYPE, ELEMENT_TYPE, NUM_ELEMENTS)      \
void elementwise_add_##INPUT_TYPE(torch::Tensor a, torch::Tensor b, torch::Tensor c){       \
    CHECK_TORCH_TYPE(a, TORCH_TYPE);                                                        \
    CHECK_TORCH_TYPE(b, TORCH_TYPE);                                                        \
    const int n = a.dim();                                                                  \
    int N = 1;                                                                              \
    for(int i = 0; i < n; i++){                                                             \
        N *= a.size(i);                                                                     \
    }                                                                                       \
    if(n == 2){                                                                             \
        int s = a.size(0);                                                                  \
        int k = a.size(1);                                                                  \
        if(k / NUM_ELEMENTS <= 1024){                                                       \
            dim3 block(k / NUM_ELEMENTS);                                                   \
            dim3 grid(s);                                                                   \
            elementwise_add_kernel_##INPUT_TYPE<<<grid, block>>>(         \
                reinterpret_cast<ELEMENT_TYPE*>(a.data_ptr()),                              \
                reinterpret_cast<ELEMENT_TYPE*>(b.data_ptr()),                              \
                reinterpret_cast<ELEMENT_TYPE*>(c.data_ptr()),                              \
                N                                                                           \
            );                                                                              \
        }                                                                                   \
        else{                                                                               \
            dim3 block(256 / NUM_ELEMENTS);                                                 \
            dim3 grid((N + 256 - 1) / 256);                                                 \
            elementwise_add_kernel_##INPUT_TYPE<<<grid, block>>>(       \
                reinterpret_cast<ELEMENT_TYPE*>(a.data_ptr()),                              \
                reinterpret_cast<ELEMENT_TYPE*>(b.data_ptr()),                              \
                reinterpret_cast<ELEMENT_TYPE*>(c.data_ptr()),                              \
                N                                                                           \
            );                                                                              \
            }                                                                               \
        }                                                                                   \
    else{                                                                                   \
        dim3 block(256 / NUM_ELEMENTS);                                                     \
        dim3 grid((N + 256 - 1) / 256);                                                     \
        elementwise_add_kernel_##INPUT_TYPE<<<grid, block>>>(           \
            reinterpret_cast<ELEMENT_TYPE*>(a.data_ptr()),                                  \
            reinterpret_cast<ELEMENT_TYPE*>(b.data_ptr()),                                  \
            reinterpret_cast<ELEMENT_TYPE*>(c.data_ptr()),                                  \
            N                                                                               \
        );                                                                                  \
    }                                                                                       \
}

ELEMENT_WISE_TORCH_BINDING(f16, torch::kHalf, half, 1)
ELEMENT_WISE_TORCH_BINDING(f16x2, torch::kHalf, half, 2)
ELEMENT_WISE_TORCH_BINDING(f16x8, torch::kHalf, half, 8)


#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
m.def(STRINGFY(func), func);


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    TORCH_BINDING_COMMON_EXTENSION(elementwise_add_f16)
    TORCH_BINDING_COMMON_EXTENSION(elementwise_add_f16x2)
    TORCH_BINDING_COMMON_EXTENSION(elementwise_add_f16x8)

}