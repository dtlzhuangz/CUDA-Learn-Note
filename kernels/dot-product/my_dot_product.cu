#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
// #define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&value)[0])
#define HALF2(value) (reinterpret_cast<__half2*>(&value)[0])
#define LD128(value) (reinterpret_cast<float4*>(&value)[0])


#define CHECK_TORCH_TYPE(tensor, type)                      \
if(tensor.options().dtype() != type){                       \
    throw std::runtime_error("tensor type must be"#type);   \
}

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f16_f32(half val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return __half2float(val);
}

template<const int THREAD_NUM>
__global__ void dot_prod_kernel_f32_f32(float* a, float* b, float* c, int N){
    __shared__ float shm[THREAD_NUM];
    int tid = threadIdx.x + blockIdx.x * THREAD_NUM;
    float prod = 0.0;
    if(tid < N)
        prod = a[tid] * b[tid];

    int lane = threadIdx.x % 32;
    int warp_idx = threadIdx.x / 32;
    if(tid < N){
        shm[warp_idx] = warp_reduce_sum_f32(prod);
    }
    __syncthreads();
    
    float t = threadIdx.x < THREAD_NUM / WARP_SIZE ? shm[lane] : 0.0f;
    if(warp_idx == 0)
        t = warp_reduce_sum_f32(t);

    if(threadIdx.x == 0)
        atomicAdd(c, t);
}

template<const int THREAD_NUM>
__global__ void dot_prod_kernel_f32_f32x4(float* a, float* b, float* c, int N){
    __shared__ float shm[THREAD_NUM];
    int tid = (threadIdx.x + blockIdx.x * THREAD_NUM) * 4;
    float prod = 0.0;
    float4 reg_a = FLOAT4(a[tid]);
    float4 reg_b = FLOAT4(b[tid]);
    if(tid < N)
        prod = reg_a.x * reg_b.x + reg_a.y * reg_b.y + reg_a.z * reg_b.z + reg_a.w * reg_b.w;

    int lane = threadIdx.x % 32;
    int warp_idx = threadIdx.x / 32;
    if(tid < N){
        shm[warp_idx] = warp_reduce_sum_f32(prod);
    }
    __syncthreads();
    
    float t = threadIdx.x < THREAD_NUM / WARP_SIZE ? shm[lane] : 0.0f;
    if(warp_idx == 0)
        t = warp_reduce_sum_f32(t);

    if(threadIdx.x == 0)
        atomicAdd(c, t);
}

template<const int THREAD_NUM>
__global__ void dot_prod_kernel_f16_f16(half* a, half* b, float* c, int N){
    __shared__ float shm[THREAD_NUM / WARP_SIZE];
    int tid = threadIdx.x + blockIdx.x * THREAD_NUM;
    float prod = 0.0;
    if(tid < N)
        prod = a[tid] * b[tid];

    int lane = threadIdx.x % 32;
    int warp_idx = threadIdx.x / 32;
    if(tid < N){
        shm[warp_idx] = warp_reduce_sum_f16_f32(prod);
    }
    __syncthreads();
    
    float t = threadIdx.x < THREAD_NUM / WARP_SIZE ? shm[lane] : 0.0f;
    if(warp_idx == 0)
        t = warp_reduce_sum_f32(t);

    if(threadIdx.x == 0)
        atomicAdd(c, t);
}

template<const int THREAD_NUM>
__global__ void dot_prod_kernel_f16_f16x8(half* a, half* b, float* c, int N){
    __shared__ float shm[THREAD_NUM / WARP_SIZE];
    int tid = (threadIdx.x + blockIdx.x * THREAD_NUM) * 8;
    const half z = __float2half(0.0);
    half prod = z;
    half reg_A[8], reg_B[8];
    LD128(reg_A[0]) = LD128(a[tid]);
    LD128(reg_B[0]) = LD128(b[tid]);

    #pragma unroll 
    for(int i = 0; i < 8; i += 2){
        half2 v = __hmul2(HALF2(reg_A[i]), HALF2(reg_B[i]));
        prod += (i + tid < N ? v.x + v.y : z);
    }
    

    int lane = threadIdx.x % 32;
    int warp_idx = threadIdx.x / 32;
    if(tid < N){
        shm[warp_idx] = warp_reduce_sum_f16_f32(prod);
    }
    __syncthreads();

    float t = threadIdx.x < THREAD_NUM / WARP_SIZE ? shm[lane] : 0.0f;
    // if(threadIdx.x < 32) printf("%d %d %f\n", blockIdx.x, threadIdx.x, t);
    if(warp_idx == 0)
        t = warp_reduce_sum_f32(t);
    if(threadIdx.x == 0)
        atomicAdd(c, t);
}


#define TORCH_BINDING_DOT_PROD(TYPE_INPUT, TYPE_ACT, TYPE_TORCH, ELEMENT_TYPE, ACCESS_NUM)   \
torch::Tensor dot_prod_##TYPE_INPUT##_##TYPE_ACT(torch::Tensor a, torch::Tensor b){             \
    CHECK_TORCH_TYPE(a, TYPE_TORCH); \
    CHECK_TORCH_TYPE(b, TYPE_TORCH); \
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device( \
    torch::kCUDA, 0); \
    auto prod = torch::zeros({1}, options); \
    const int n = a.dim(); \
    int N = 1; \
    for(int i = 0; i < n; i++){ \
        N *= a.size(i); \
    } \
    dim3 block(256); \
    dim3 grid((N + 256 - 1) / 256 / ACCESS_NUM); \
    dot_prod_kernel_##TYPE_INPUT##_##TYPE_ACT<256><<<grid, block>>>( \
        reinterpret_cast<ELEMENT_TYPE*>(a.data_ptr()), \
        reinterpret_cast<ELEMENT_TYPE*>(b.data_ptr()), \
        reinterpret_cast<float*>(prod.data_ptr()), \
        N \
    ); \
    return prod; \
}

TORCH_BINDING_DOT_PROD(f32,        f32,  torch::kFloat32,       float,              1)
TORCH_BINDING_DOT_PROD(f32,        f32x4,  torch::kFloat32,       float,              4)
TORCH_BINDING_DOT_PROD(f16,        f16,  torch::kHalf,       half,              1)
TORCH_BINDING_DOT_PROD(f16,        f16x8,  torch::kHalf,       half,              8)


#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
m.def(STRINGFY(func), func);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(dot_prod_f32_f32)
  TORCH_BINDING_COMMON_EXTENSION(dot_prod_f32_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(dot_prod_f16_f16)
  TORCH_BINDING_COMMON_EXTENSION(dot_prod_f16_f16x8)
}